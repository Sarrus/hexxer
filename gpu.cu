
#include <hip/hip_runtime.h>
#include <iostream>

#define HEXAGON_AS_INT u_int64_t
#define TOTAL_SEGMENTS 19
#define TOTAL_SEGMENTS_WITH_LEFT_RED_LOCKED 18
#define LOCKED_RED_LOCATION 18

enum colours{
    RED,
    YELLOW,
    GREEN,
    BLUE
};

__device__
const struct {
    char countOfRelationships;
    char relatedCells[6];
} cellRelationships[TOTAL_SEGMENTS] = {
        {3, {1, 3, 4}}, // 0
        {4, {0, 2, 4, 5}}, // 1
        {3, {1, 5, 6}}, // 2
        {4, {0, 4, 7, 8}}, // 3
        {6, {0, 1, 3,5,8,9}}, // 4
        {6, {1, 2, 4, 6, 9, 10}}, // 5
        {4, {2, 5, 10, 11}}, // 6
        {3, {3, 8, 12}}, // 7
        {6, {3, 4, 7, 9, 12, 13}}, // 8
        {6, {4, 5, 8, 10, 13, 14}}, // 9
        {6, {5, 6, 9, 11, 14, 15}}, // 10
        {3, {6, 10, 15}}, // 11
        {4, {7, 8, 13, 16}}, // 12
        {6, {8, 9, 12, 14, 16, 17}}, // 13
        {6, {9, 10, 13, 15, 17, 18}}, // 14
        {4, {10, 11, 14, 18}}, // 15
        {3, {12, 13, 17}}, // 16
        {4, {13, 14, 16, 18}}, // 17
        {3, {14, 15, 17}}
};

__device__
bool validateSolution(HEXAGON_AS_INT solution)
{
    char reds = 0;
    char yellows = 0;
    char greens = 0;
    char blues = 0;

    char redsAt[TOTAL_SEGMENTS];
    char yellowsAt[TOTAL_SEGMENTS];
    char greensAt[TOTAL_SEGMENTS];
    char bluesAt[TOTAL_SEGMENTS];

    // Count the colours used and record their locations. We don't count the last cell yet because it is not included
    // in the first validation stage.
    for(char i = 0; i < TOTAL_SEGMENTS_WITH_LEFT_RED_LOCKED; i++)
    {
        switch((solution >> i * 2) % 4)
        {
            case RED:
                redsAt[reds] = i;
                reds++;
                break;

            case YELLOW:
                yellowsAt[yellows] = i;
                yellows++;
                break;

            case GREEN:
                greensAt[greens] = i;
                greens++;
                break;

            case BLUE:
                bluesAt[blues] = i;
                blues++;
                break;
        }
    }

    if(
            (reds < 3)
            || (yellows < 3)
            || (greens < 3)
            || (blues < 3)
            )
    {
        // Less than three of a colour found
        return false;
    }

    // Now we add the last cell to the tally.
    redsAt[reds] = LOCKED_RED_LOCATION;
    reds++;

    for(char i = 0; i < greens; i++)
    {
        char redsAroundGreen = 0;

        for(char j = 0; j < cellRelationships[greensAt[i]].countOfRelationships; j++)
        {
            for(char k = 0; k < reds; k++)
            {
                if(redsAt[k] == cellRelationships[greensAt[i]].relatedCells[j])
                {
                    redsAroundGreen++;
                }
            }
        }

        if(redsAroundGreen != 3)
        {
            // green not surrounded by three reds found
            return false;
        }
    }

    for(char i = 0; i < blues; i++)
    {
        char yellowsAroundBlue = 0;

        for(char j = 0; j < cellRelationships[bluesAt[i]].countOfRelationships; j++)
        {
            for(char k = 0; k < yellows; k++)
            {
                if(yellowsAt[k] == cellRelationships[bluesAt[i]].relatedCells[j])
                {
                    yellowsAroundBlue++;
                }
            }
        }

        if(yellowsAroundBlue != 2)
        {
            // Blue not surrounded by two yellows found
            return false;
        }
    }

    for(char i = 0; i < yellows; i++)
    {
        char coloursFoundSurrounding = 0b000; // 0b001 = red, 0b010 = green, 0b100 = blue

        for(char j = 0; j < cellRelationships[yellowsAt[i]].countOfRelationships; j++)
        {
            for(char k = 0; k < reds; k++)
            {
                if(redsAt[k] == cellRelationships[yellowsAt[i]].relatedCells[j])
                {
                    coloursFoundSurrounding |= 0b001;
                    k = reds;
                }
            }

            for(char k = 0; k < greens; k++)
            {
                if(greensAt[k] == cellRelationships[yellowsAt[i]].relatedCells[j])
                {
                    coloursFoundSurrounding |= 0b010;
                    k = greens;
                }
            }

            for(char k = 0; k < blues; k++)
            {
                if(bluesAt[k] == cellRelationships[yellowsAt[i]].relatedCells[j])
                {
                    coloursFoundSurrounding |= 0b100;
                    k = blues;
                }
            }
        }

        if(coloursFoundSurrounding != 0b111)
        {
            // Yellow not surrounded by at least one of every other colour found
            return false;
        }
    }

    return true;
}

__global__
void solver(bool * result)
{
    *result = validateSolution(152953033);
}

extern "C" void solveWithCUDA()
{
    bool * result;
    hipMallocManaged(&result, sizeof(bool));
    solver<<<1, 1>>>(result);
    hipDeviceSynchronize();
    if(*result)
    {
        std::cerr << "Valid Solution" << std::endl;
    }
    else
    {
        std::cerr << "Invalid Solution" << std::endl;
    }
}