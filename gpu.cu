#include "hip/hip_runtime.h"
#include <iostream>
#include <unistd.h>
#include "memory.cuh"
#include "hexagon.cuh"
#include "configuration.cuh"

#define THREAD_COUNT 80 * 256
#define FOUND_SOLUTION_POOL_SIZE 10000

__device__
const struct {
    char countOfRelationships;
    char relatedCells[6];
} cellRelationships[TOTAL_SEGMENTS] = {
        {3, {1, 3, 4}}, // 0
        {4, {0, 2, 4, 5}}, // 1
        {3, {1, 5, 6}}, // 2
        {4, {0, 4, 7, 8}}, // 3
        {6, {0, 1, 3,5,8,9}}, // 4
        {6, {1, 2, 4, 6, 9, 10}}, // 5
        {4, {2, 5, 10, 11}}, // 6
        {3, {3, 8, 12}}, // 7
        {6, {3, 4, 7, 9, 12, 13}}, // 8
        {6, {4, 5, 8, 10, 13, 14}}, // 9
        {6, {5, 6, 9, 11, 14, 15}}, // 10
        {3, {6, 10, 15}}, // 11
        {4, {7, 8, 13, 16}}, // 12
        {6, {8, 9, 12, 14, 16, 17}}, // 13
        {6, {9, 10, 13, 15, 17, 18}}, // 14
        {4, {10, 11, 14, 18}}, // 15
        {3, {12, 13, 17}}, // 16
        {4, {13, 14, 16, 18}}, // 17
        {3, {14, 15, 17}}
};

__device__ HEXAGON_AS_INT threadFoundSolutions[THREAD_COUNT];
__device__ HEXAGON_AS_INT threadTriedSolutions[THREAD_COUNT];
__device__ HEXAGON_AS_INT threadFoundSolutionPool[THREAD_COUNT][FOUND_SOLUTION_POOL_SIZE];
__device__ HEXAGON_AS_INT threadFoundSolutionsLastPulledTo[THREAD_COUNT];

__device__
bool validateSolution(HEXAGON_AS_INT solution)
{
    char reds = 0;
    char yellows = 0;
    char greens = 0;
    char blues = 0;

    char redsAt[TOTAL_SEGMENTS];
    char yellowsAt[TOTAL_SEGMENTS];
    char greensAt[TOTAL_SEGMENTS];
    char bluesAt[TOTAL_SEGMENTS];

    // Count the colours used and record their locations. We don't count the last cell yet because it is not included
    // in the first validation stage.
    for(char i = 0; i < TOTAL_SEGMENTS_WITH_LEFT_RED_LOCKED; i++)
    {
        switch((solution >> i * 2) % 4)
        {
            case RED:
                redsAt[reds] = i;
                reds++;
                break;

            case YELLOW:
                yellowsAt[yellows] = i;
                yellows++;
                break;

            case GREEN:
                greensAt[greens] = i;
                greens++;
                break;

            case BLUE:
                bluesAt[blues] = i;
                blues++;
                break;
        }
    }

    if(
            (reds < 3)
            || (yellows < 3)
            || (greens < 3)
            || (blues < 3)
            )
    {
        // Less than three of a colour found
        return false;
    }

    // Now we add the last cell to the tally.
    redsAt[reds] = LOCKED_RED_LOCATION;
    reds++;

    for(char i = 0; i < greens; i++)
    {
        char redsAroundGreen = 0;

        for(char j = 0; j < cellRelationships[greensAt[i]].countOfRelationships; j++)
        {
            for(char k = 0; k < reds; k++)
            {
                if(redsAt[k] == cellRelationships[greensAt[i]].relatedCells[j])
                {
                    redsAroundGreen++;
                }
            }
        }

        if(redsAroundGreen != 3)
        {
            // green not surrounded by three reds found
            return false;
        }
    }

    for(char i = 0; i < blues; i++)
    {
        char yellowsAroundBlue = 0;

        for(char j = 0; j < cellRelationships[bluesAt[i]].countOfRelationships; j++)
        {
            for(char k = 0; k < yellows; k++)
            {
                if(yellowsAt[k] == cellRelationships[bluesAt[i]].relatedCells[j])
                {
                    yellowsAroundBlue++;
                }
            }
        }

        if(yellowsAroundBlue != 2)
        {
            // Blue not surrounded by two yellows found
            return false;
        }
    }

    for(char i = 0; i < yellows; i++)
    {
        char coloursFoundSurrounding = 0b000; // 0b001 = red, 0b010 = green, 0b100 = blue

        for(char j = 0; j < cellRelationships[yellowsAt[i]].countOfRelationships; j++)
        {
            for(char k = 0; k < reds; k++)
            {
                if(redsAt[k] == cellRelationships[yellowsAt[i]].relatedCells[j])
                {
                    coloursFoundSurrounding |= 0b001;
                    k = reds;
                }
            }

            for(char k = 0; k < greens; k++)
            {
                if(greensAt[k] == cellRelationships[yellowsAt[i]].relatedCells[j])
                {
                    coloursFoundSurrounding |= 0b010;
                    k = greens;
                }
            }

            for(char k = 0; k < blues; k++)
            {
                if(bluesAt[k] == cellRelationships[yellowsAt[i]].relatedCells[j])
                {
                    coloursFoundSurrounding |= 0b100;
                    k = blues;
                }
            }
        }

        if(coloursFoundSurrounding != 0b111)
        {
            // Yellow not surrounded by at least one of every other colour found
            return false;
        }
    }

    return true;
}

__global__
void prepare()
{
    for(int i = 0; i < THREAD_COUNT; i++)
    {
        threadFoundSolutions[i] = 0;
        threadTriedSolutions[i] = 0;
        threadFoundSolutionsLastPulledTo[i] = 0;
    }
}

__global__
void solver(HEXAGON_AS_INT * aValidSolution, const bool * kernelStop)
{
    HEXAGON_AS_INT start = blockIdx.x * blockDim.x + threadIdx.x;
    HEXAGON_AS_INT step = blockDim.x * gridDim.x;

    for(HEXAGON_AS_INT i = start; i < TOTAL_HEXAGONS_WITH_LEFT_RED_LOCKED; i += step)
    {
        if(validateSolution(i))
        {
            //printf("%lu\r\n", i);
            threadFoundSolutionPool[start][threadFoundSolutions[start]] = i;
            threadFoundSolutions[start]++;
            *aValidSolution = i;
        }
        threadTriedSolutions[start]++;

        if(*kernelStop)
        {
            return;
        }
    }
}

__global__
void retrieveResult(HEXAGON_AS_INT * solutionCount)
{
    *solutionCount = 0;
    for(int i = 0; i < THREAD_COUNT; i++)
    {
        *solutionCount += threadFoundSolutions[i];
    }
}

__global__
void manageProgress(HEXAGON_AS_INT * triedSolutions, HEXAGON_AS_INT * foundSolutions, HEXAGON_AS_INT * solutionCount)
{
    *triedSolutions = 0;
    for(HEXAGON_AS_INT i = 0; i < THREAD_COUNT; i++)
    {
        *triedSolutions += threadTriedSolutions[i];
        while(threadFoundSolutionsLastPulledTo[i] < threadFoundSolutions[i])
        {
            foundSolutions[*solutionCount] = threadFoundSolutionPool[i][threadFoundSolutionsLastPulledTo[i]];
            (*solutionCount)++;
            threadFoundSolutionsLastPulledTo[i]++;
        }
    }
}

extern "C" void solveWithCUDA()
{
    HEXAGON_AS_INT * solutionCount;
    HEXAGON_AS_INT * aValidSolution;
    HEXAGON_AS_INT * triedSolutions;
    HEXAGON_AS_INT * foundSolutions;
    bool * kernelStop;
    hipMallocManaged(&solutionCount, sizeof(HEXAGON_AS_INT));
    hipMallocManaged(&aValidSolution, sizeof(HEXAGON_AS_INT));
    hipMallocManaged(&triedSolutions, sizeof(HEXAGON_AS_INT));
    hipMallocManaged(&foundSolutions, sizeof(HEXAGON_AS_INT) * FOUND_SOLUTION_POOL_SIZE);
    hipMallocManaged(&kernelStop, sizeof(bool));

    char solutionString[ID_PRINT_MAX_SIZE];

    hipStream_t mainStream, verificationStream;
    hipStreamCreate(&mainStream);
    hipStreamCreate(&verificationStream);

    *kernelStop = false;

    prepare<<<1, 1, 0, mainStream>>>();
    solver<<<80, 256, 0, mainStream>>>(aValidSolution, kernelStop);
    //retrieveResult<<<1, 1, 0, mainStream>>>(solutionCount);

    HEXAGON_AS_INT solutionsPushed = 0;
    HEXAGON_AS_INT uniqueSolutions = 0;

    bool looping = true;
    bool lastPrintWasProgressLine = false;

    while(looping)
    {
        if(hipStreamQuery(mainStream) == hipSuccess)
        {
            looping = false;
        }
        manageProgress<<<1, 1, 0, verificationStream>>>(triedSolutions, foundSolutions, solutionCount);
        hipStreamSynchronize(verificationStream);
        while(solutionsPushed < *solutionCount)
        {
            if(lastPrintWasProgressLine)
            {
                fprintf(stderr, "%c[2K\r", 27);
                lastPrintWasProgressLine = false;
            }
            fprintf(stderr, "Solution found at hexagon no. %lu, ", foundSolutions[solutionsPushed]);

            if(saveAllSolutionIDs)
            {
                snprintf(solutionString, ID_PRINT_MAX_SIZE, "%lu\r\n", foundSolutions[solutionsPushed]);
                write(allSolutionsLocationHandle, solutionString, strlen(solutionString));
            }

            HEXAGON_AS_INT matchedSolution = checkSolutionForVisualMatches(foundSolutions[solutionsPushed]);
            if(stopOnFirstSolution)
            {
                fprintf(stderr, "stopping.\r\n");
                *kernelStop = true;
                looping = false;
                break;
            }
            else if(matchedSolution)
            {
                fprintf(stderr, "visually matches solution no. %lu ", matchedSolution);
            }
            else
            {
                uniqueSolutions++;
                fprintf(stderr, "no visual matches found. ");
                if(saveUniqueSolutionIDs)
                {
                    snprintf(solutionString, ID_PRINT_MAX_SIZE, "%lu\r\n", foundSolutions[solutionsPushed]);
                    write(uniqueSolutionsLocationHandle, solutionString, strlen(solutionString));
                }
            }
            fprintf(stderr, "%lu solutions found so far, %lu visually unique.\r\n", solutionsPushed, uniqueSolutions);

            storeSolution(foundSolutions[solutionsPushed]);
            solutionsPushed++;
        }

        if(lastPrintWasProgressLine)
        {
            fprintf(stderr, "%c[2K\r", 27);
        }
        fprintf(stderr,
                "%lu hexagons processed so far, %f%% of total.",
                *triedSolutions,
                100 * (float)*triedSolutions / (float)TOTAL_HEXAGONS_WITH_LEFT_RED_LOCKED
        );
        lastPrintWasProgressLine = true;

        sleep(1);
    }

    hipDeviceSynchronize();
    if(stopOnFirstSolution)
    {
        fprintf(stderr, "\r\nStopped.\r\n");
    }
    else
    {
        fprintf(stderr, "\r\nTried all possible solutions.\r\n");
    }
}