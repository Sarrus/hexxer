
#include <hip/hip_runtime.h>
#include <iostream>
#include <unistd.h>

#define HEXAGON_AS_INT u_int64_t
#define TOTAL_SEGMENTS 19
#define TOTAL_SEGMENTS_WITH_LEFT_RED_LOCKED 18
#define LOCKED_RED_LOCATION 18
#define TOTAL_HEXAGONS_WITH_LEFT_RED_LOCKED 0x1000000000
#define THREAD_COUNT 80 * 256
#define FOUND_SOLUTION_POOL_SIZE 10000

enum colours{
    RED,
    YELLOW,
    GREEN,
    BLUE
};

__device__
const struct {
    char countOfRelationships;
    char relatedCells[6];
} cellRelationships[TOTAL_SEGMENTS] = {
        {3, {1, 3, 4}}, // 0
        {4, {0, 2, 4, 5}}, // 1
        {3, {1, 5, 6}}, // 2
        {4, {0, 4, 7, 8}}, // 3
        {6, {0, 1, 3,5,8,9}}, // 4
        {6, {1, 2, 4, 6, 9, 10}}, // 5
        {4, {2, 5, 10, 11}}, // 6
        {3, {3, 8, 12}}, // 7
        {6, {3, 4, 7, 9, 12, 13}}, // 8
        {6, {4, 5, 8, 10, 13, 14}}, // 9
        {6, {5, 6, 9, 11, 14, 15}}, // 10
        {3, {6, 10, 15}}, // 11
        {4, {7, 8, 13, 16}}, // 12
        {6, {8, 9, 12, 14, 16, 17}}, // 13
        {6, {9, 10, 13, 15, 17, 18}}, // 14
        {4, {10, 11, 14, 18}}, // 15
        {3, {12, 13, 17}}, // 16
        {4, {13, 14, 16, 18}}, // 17
        {3, {14, 15, 17}}
};

__device__ HEXAGON_AS_INT threadFoundSolutions[THREAD_COUNT];
__device__ HEXAGON_AS_INT threadTriedSolutions[THREAD_COUNT];
__device__ HEXAGON_AS_INT threadFoundSolutionPool[THREAD_COUNT][FOUND_SOLUTION_POOL_SIZE];
__device__ HEXAGON_AS_INT threadFoundSolutionsLastPulledTo[THREAD_COUNT];

__device__
bool validateSolution(HEXAGON_AS_INT solution)
{
    char reds = 0;
    char yellows = 0;
    char greens = 0;
    char blues = 0;

    char redsAt[TOTAL_SEGMENTS];
    char yellowsAt[TOTAL_SEGMENTS];
    char greensAt[TOTAL_SEGMENTS];
    char bluesAt[TOTAL_SEGMENTS];

    // Count the colours used and record their locations. We don't count the last cell yet because it is not included
    // in the first validation stage.
    for(char i = 0; i < TOTAL_SEGMENTS_WITH_LEFT_RED_LOCKED; i++)
    {
        switch((solution >> i * 2) % 4)
        {
            case RED:
                redsAt[reds] = i;
                reds++;
                break;

            case YELLOW:
                yellowsAt[yellows] = i;
                yellows++;
                break;

            case GREEN:
                greensAt[greens] = i;
                greens++;
                break;

            case BLUE:
                bluesAt[blues] = i;
                blues++;
                break;
        }
    }

    if(
            (reds < 3)
            || (yellows < 3)
            || (greens < 3)
            || (blues < 3)
            )
    {
        // Less than three of a colour found
        return false;
    }

    // Now we add the last cell to the tally.
    redsAt[reds] = LOCKED_RED_LOCATION;
    reds++;

    for(char i = 0; i < greens; i++)
    {
        char redsAroundGreen = 0;

        for(char j = 0; j < cellRelationships[greensAt[i]].countOfRelationships; j++)
        {
            for(char k = 0; k < reds; k++)
            {
                if(redsAt[k] == cellRelationships[greensAt[i]].relatedCells[j])
                {
                    redsAroundGreen++;
                }
            }
        }

        if(redsAroundGreen != 3)
        {
            // green not surrounded by three reds found
            return false;
        }
    }

    for(char i = 0; i < blues; i++)
    {
        char yellowsAroundBlue = 0;

        for(char j = 0; j < cellRelationships[bluesAt[i]].countOfRelationships; j++)
        {
            for(char k = 0; k < yellows; k++)
            {
                if(yellowsAt[k] == cellRelationships[bluesAt[i]].relatedCells[j])
                {
                    yellowsAroundBlue++;
                }
            }
        }

        if(yellowsAroundBlue != 2)
        {
            // Blue not surrounded by two yellows found
            return false;
        }
    }

    for(char i = 0; i < yellows; i++)
    {
        char coloursFoundSurrounding = 0b000; // 0b001 = red, 0b010 = green, 0b100 = blue

        for(char j = 0; j < cellRelationships[yellowsAt[i]].countOfRelationships; j++)
        {
            for(char k = 0; k < reds; k++)
            {
                if(redsAt[k] == cellRelationships[yellowsAt[i]].relatedCells[j])
                {
                    coloursFoundSurrounding |= 0b001;
                    k = reds;
                }
            }

            for(char k = 0; k < greens; k++)
            {
                if(greensAt[k] == cellRelationships[yellowsAt[i]].relatedCells[j])
                {
                    coloursFoundSurrounding |= 0b010;
                    k = greens;
                }
            }

            for(char k = 0; k < blues; k++)
            {
                if(bluesAt[k] == cellRelationships[yellowsAt[i]].relatedCells[j])
                {
                    coloursFoundSurrounding |= 0b100;
                    k = blues;
                }
            }
        }

        if(coloursFoundSurrounding != 0b111)
        {
            // Yellow not surrounded by at least one of every other colour found
            return false;
        }
    }

    return true;
}

__global__
void prepare()
{
    for(int i = 0; i < THREAD_COUNT; i++)
    {
        threadFoundSolutions[i] = 0;
        threadTriedSolutions[i] = 0;
        threadFoundSolutionsLastPulledTo[i] = 0;
    }
}

__global__
void solver(HEXAGON_AS_INT * aValidSolution)
{
    HEXAGON_AS_INT start = blockIdx.x * blockDim.x + threadIdx.x;
    HEXAGON_AS_INT step = blockDim.x * gridDim.x;

    for(HEXAGON_AS_INT i = start; i < TOTAL_HEXAGONS_WITH_LEFT_RED_LOCKED; i += step)
    {
        if(validateSolution(i))
        {
            //printf("%lu\r\n", i);
            threadFoundSolutionPool[start][threadFoundSolutions[start]] = i;
            threadFoundSolutions[start]++;
            *aValidSolution = i;
        }
        threadTriedSolutions[start]++;
    }
}

__global__
void retrieveResult(HEXAGON_AS_INT * solutionCount)
{
    *solutionCount = 0;
    for(int i = 0; i < THREAD_COUNT; i++)
    {
        *solutionCount += threadFoundSolutions[i];
    }
}

__global__
void manageProgress(HEXAGON_AS_INT * triedSolutions, HEXAGON_AS_INT * foundSolutions, HEXAGON_AS_INT * solutionCount)
{
    *triedSolutions = 0;
    for(HEXAGON_AS_INT i = 0; i < THREAD_COUNT; i++)
    {
        *triedSolutions += threadTriedSolutions[i];
        while(threadFoundSolutionsLastPulledTo[i] < threadFoundSolutions[i])
        {
            foundSolutions[*solutionCount] = threadFoundSolutionPool[i][threadFoundSolutionsLastPulledTo[i]];
            (*solutionCount)++;
            threadFoundSolutionsLastPulledTo[i]++;
        }
    }
}

extern "C" void solveWithCUDA()
{
    HEXAGON_AS_INT * solutionCount;
    HEXAGON_AS_INT * aValidSolution;
    HEXAGON_AS_INT * triedSolutions;
    HEXAGON_AS_INT * foundSolutions;
    hipMallocManaged(&solutionCount, sizeof(HEXAGON_AS_INT));
    hipMallocManaged(&aValidSolution, sizeof(HEXAGON_AS_INT));
    hipMallocManaged(&triedSolutions, sizeof(HEXAGON_AS_INT));
    hipMallocManaged(&foundSolutions, sizeof(HEXAGON_AS_INT) * FOUND_SOLUTION_POOL_SIZE);

    hipStream_t mainStream, verificationStream;
    hipStreamCreate(&mainStream);
    hipStreamCreate(&verificationStream);

    prepare<<<1, 1, 0, mainStream>>>();
    solver<<<80, 256, 0, mainStream>>>(aValidSolution);
    //retrieveResult<<<1, 1, 0, mainStream>>>(solutionCount);

    HEXAGON_AS_INT solutionsPrinted = 0;

    while(hipStreamQuery(mainStream) != hipSuccess)
    {
        manageProgress<<<1, 1, 0, verificationStream>>>(triedSolutions, foundSolutions, solutionCount);
        hipStreamSynchronize(verificationStream);
        fprintf(stderr,
                "%lu hexagons processed so far, %f%% of total.\r\n",
                *triedSolutions,
                100 * (float)*triedSolutions / (float)TOTAL_HEXAGONS_WITH_LEFT_RED_LOCKED
        );
        while(solutionsPrinted < *solutionCount)
        {
            fprintf(stderr, "New solution: %lu\r\n", foundSolutions[solutionsPrinted]);
            solutionsPrinted++;
        }
        sleep(1);
    }

    hipDeviceSynchronize();
    std::cerr << "Solutions found: " << *solutionCount << std::endl;
    std::cerr << "A valid solution: " << *aValidSolution << std::endl;
}